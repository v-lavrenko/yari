#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//#include <cuda.h>
#include <hip/hip_runtime.h>

extern "C" {
void show_slice (float *S, uint nr, uint nc, const char *tag) ;
void slice_to_mtx (void *M, uint r0, uint c0, uint nr, uint nc, float *S) ;
void mtx_to_slice (void *M, uint r0, uint c0, uint nr, uint nc, float *S) ;
void *open_coll (char *path, const char *access) ;
void free_coll (void *P);
uint num_rows (void *rows) ;
uint num_cols (void *rows) ;
}

// see /usr/local/cuda-11.8/targets/x86_64-linux/include/driver_types.h
size_t gpu_ram (int dev) {
  hipSetDevice(dev);
  struct hipDeviceProp_t prm;
  hipGetDeviceProperties(&prm, dev);  
  size_t bytes = prm.totalGlobalMem;
  double GB = (bytes>>20) / 1024.0;
  fprintf(stderr,"%.2fG VRAM on %s\n", GB, prm.name);
  return bytes;
}

// AC [a,c] = SUM_b AB[a,b] * BC[b,c] ... assume one thread per cell of AxC
__global__ void product1 (float *AC, uint na, float *AB, uint nb, float *BC, uint nc) {
  uint a = blockIdx.x * blockDim.x + threadIdx.x, b; // x is the row
  uint c = blockIdx.y * blockDim.y + threadIdx.y;    // y is the col
  if (a >= na || c >= nc) return;
  double result = 0;
  for (b = 0; b < nb; ++b) 
    result += AB[a*nb + b] * BC[b*nc + c]; // += AB[a,b] * BC[b,c]
  AC[a*nc + c] = result; // 'a' rows of length 'nc', then 'c' elements
}

__global__ void set (float *AC, uint na, float *AB, uint nb, float *BC, uint nc) {
  uint a = blockIdx.x * blockDim.x + threadIdx.x; // x is the row
  uint c = blockIdx.y * blockDim.y + threadIdx.y; // y is the col
  if (a >= na || c >= nc) return;
  AC[a*nc + c] = AB[a*nb + c] * BC[a*nb + c]; 
}


// [AxC] += [AxB] x [BxC]
void _product (void *AxC, void *AxB, void *BxC) {
  assert (num_cols(AxB) == num_rows(BxC)); 
  uint nA = num_rows(AxB), nB = num_rows (BxC), nC = num_cols (BxC);
  size_t vram = gpu_ram (0), sz = sizeof(float);
  assert ((nA * nC + nA * nB + nB * nC) * sz < vram); // ensure vram
  float *AC, *AB, *BC;
  hipMallocManaged(&AC, nA * nC * sz);
  hipMallocManaged(&AB, nA * nB * sz);
  hipMallocManaged(&BC, nB * nC * sz);
  
  uint na = nA, nb = nB, nc = nC;
  
  //                     r0 c0 nr  nc
  mtx_to_slice (AxB, 0, 0, na, nb, AB);
  mtx_to_slice (BxC, 0, 0, nb, nc, BC);
  
  dim3 block(32,32); // max threads per block is 2014
  dim3 grid(na/32, nc/32); // [na x nc] total threads, one per cell AC[a,c]  
  product1 <<<grid,block>>> (AC, na, AB, nb, BC, nc);
  hipDeviceSynchronize();
  
  //show_slice (AB, na, nb, "AB");
  //show_slice (BC, nb, nc, "BC");  
  //show_slice (AC, na, nc, "AC");
  
  slice_to_mtx (AxC, 0, 0, na, nc, AC);
  
  hipFree(AC); hipFree(AB); hipFree(BC);
}

// P = A x B.T
int gpu_product (char *_P, char *_A, char *_B, char *prm) {
  void *P = open_coll (_P, "w+");
  void *A = open_coll (_A, "r+");
  void *B = open_coll (_B, "r+");
  _product (P, A, B);
  free_coll(P); free_coll(A); free_coll(B);
  return 0;
}
  
#define arg(i) ((i < argc) ? argv[i] : NULL)
#define a(i) ((i < argc) ? argv[i] : ((char*)""))

int main (int argc, char *argv[]) {
  // cumtx P = A x C [prm]
  if ((a(2)[0] == '=') && (a(4)[0] == 'x'))
    return gpu_product (arg(1), arg(3), arg(5), a(6));
  gpu_ram(0);
  return 0;
}

/*
#include <cuda_runtime.h>
#include "cublas_v2.h"
  cublasHandle_t H;
  cublasStatus_t ok, OK = CUBLAS_STATUS_SUCCESS;
  ok = cublasCreate(&H);
  if (ok != OK) { printf ("[%d] faild %s %d %d\n", ok, "cublasCreate", CUBLAS_STATUS_NOT_INITIALIZED, CUBLAS_STATUS_ALLOC_FAILED);  return 1; }
  ok = cublasGetVersion(H, &version);
  if (ok != OK) { printf ("[%d] faild %s\n", ok, "cublasVersion"); return 1; }
  ok = cublasDestroy(H);
  if (ok != OK) { printf ("[%d] faild %s\n", ok, "cublasDestroy"); return 1; }
*/
