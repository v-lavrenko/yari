#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//#include <cuda.h>
#include <hip/hip_runtime.h>

extern "C" {
  void show_slice (float *S, uint nr, uint nc, const char *tag) ;
  void slice_to_mtx (void *M, uint r0, uint c0, uint nr, uint nc, float *S) ;
  void mtx_to_slice (void *M, uint r0, uint c0, uint nr, uint nc, float *S) ;
  void mtx_to_buf (void *M, void *buf, ulong *end, uint r0, uint nr) ;  
  void *open_coll (char *path, const char *access) ;
  void free_coll (void *P);
  uint num_rows (void *rows) ;
  uint num_cols (void *rows) ;
}

// see /usr/local/cuda-11.8/targets/x86_64-linux/include/driver_types.h
size_t gpu_ram (int dev) {
  hipSetDevice(dev);
  struct hipDeviceProp_t prm;
  hipGetDeviceProperties(&prm, dev);  
  size_t bytes = prm.totalGlobalMem;
  double GB = (bytes>>20) / 1024.0;
  fprintf(stderr,"%.2fG VRAM on %s\n", GB, prm.name);
  return bytes;
}

typedef struct { unsigned i; float x; } ix_t;

__global__ void sparse_dot (float *AB, uint na, uint nb,
			    ix_t *A, ulong *aoff, 
			    ix_t *B, ulong *boff) {
  uint a = blockIdx.x * blockDim.x + threadIdx.x; // x is the row of A
  uint b = blockIdx.y * blockDim.y + threadIdx.y; // y is the row of B  
  if (a >= na || b >= nb) return; // off the grid
  ulong a0 = aoff[a], a1 = aoff[a+1]; // row a = A[a0:a1]
  ulong b0 = boff[b], b1 = boff[b+1]; // row b = B[b0:b1]
  double result = 0;
  while ((a0 < a1) && (b0 < b1)) {
    int eq = (A[a0].i == B[b0].i); // 1 iff a matches b
    int da = (A[a0].i <= B[b0].i); // 1 iff a needs to advance
    int db = (B[b0].i <= A[a0].i); // 1 iff b needs to advance
    result += eq * A[a0].x * B[b0].x;
    a0 += da;
    b0 += db;
  }
  AB[a*nb + b] = result;
}

void sparse_product (void *_C, void *_A, void *_B) {
  fprintf(stderr, "orly?\n");
  assert (num_cols(_A) == num_cols(_B)); 
  uint nA = num_rows(_A), nB = num_rows (_B), nC = num_cols (_A);
  size_t vram = gpu_ram (0);
  ix_t *A, *B;
  float *C;
  ulong *aoff, *boff;
  hipMallocManaged(&C, 3607101440); // FIXME!
  hipMallocManaged(&A, 3607101440);
  hipMallocManaged(&B, 3607101440);
  hipMallocManaged(&aoff, (nA+1)*sizeof(ulong));
  hipMallocManaged(&boff, (nB+1)*sizeof(ulong));
  
  mtx_to_buf (_A, A, aoff, 0, nA);
  mtx_to_buf (_B, B, boff, 0, nB);
  
  dim3 block(32,32); // max threads per block is 2014
  dim3 grid(nA/32, nB/32); // [nA x nB] total threads, one per cell C[a,b]
  sparse_dot <<<grid,block>>> (C, nA, nB, A, aoff, B, boff);
  hipDeviceSynchronize();
  
  //show_slice (AB, na, nb, "AB");
  //show_slice (BC, nb, nc, "BC");  
  //show_slice (AC, na, nc, "AC");
  
  slice_to_mtx (_C, 0, 0, nA, nB, C);
  
  hipFree(A); hipFree(B); hipFree(C);
  hipFree (aoff); hipFree (boff);
}


// AC [a,c] = SUM_b AB[a,b] * BC[b,c] ... assume one thread per cell of AxC, dense
__global__ void dense_dot (float *AC, uint na, float *AB, uint nb, float *BC, uint nc) {
  uint a = blockIdx.x * blockDim.x + threadIdx.x, b; // x is the row
  uint c = blockIdx.y * blockDim.y + threadIdx.y;    // y is the col
  if (a >= na || c >= nc) return;
  double result = 0;
  for (b = 0; b < nb; ++b) 
    result += AB[a*nb + b] * BC[b*nc + c]; // += AB[a,b] * BC[b,c]
  AC[a*nc + c] = result; // 'a' rows of length 'nc', then 'c' elements
}

__global__ void set (float *AC, uint na, float *AB, uint nb, float *BC, uint nc) {
  uint a = blockIdx.x * blockDim.x + threadIdx.x; // x is the row
  uint c = blockIdx.y * blockDim.y + threadIdx.y; // y is the col
  if (a >= na || c >= nc) return;
  AC[a*nc + c] = AB[a*nb + c] * BC[a*nb + c]; 
}


// [AxC] += [AxB] x [BxC]
void dense_product (void *AxC, void *AxB, void *BxC) {
  assert (num_cols(AxB) == num_rows(BxC)); 
  uint nA = num_rows(AxB), nB = num_rows (BxC), nC = num_cols (BxC);
  size_t vram = gpu_ram (0), sz = sizeof(float);
  assert ((nA * nC + nA * nB + nB * nC) * sz < vram); // ensure vram
  float *AC, *AB, *BC;
  hipMallocManaged(&AC, nA * nC * sz);
  hipMallocManaged(&AB, nA * nB * sz);
  hipMallocManaged(&BC, nB * nC * sz);
  
  uint na = nA, nb = nB, nc = nC;
  
  //                     r0 c0 nr  nc
  mtx_to_slice (AxB, 0, 0, na, nb, AB);
  mtx_to_slice (BxC, 0, 0, nb, nc, BC);
  
  dim3 block(32,32); // max threads per block is 2014
  dim3 grid(na/32, nc/32); // [na x nc] total threads, one per cell AC[a,c]  
  dense_dot <<<grid,block>>> (AC, na, AB, nb, BC, nc);
  hipDeviceSynchronize();
  
  //show_slice (AB, na, nb, "AB");
  //show_slice (BC, nb, nc, "BC");  
  //show_slice (AC, na, nc, "AC");
  
  slice_to_mtx (AxC, 0, 0, na, nc, AC);
  
  hipFree(AC); hipFree(AB); hipFree(BC);
}

// P = A x B.T
int gpu_product (char *_P, char *_A, char *_B, char *prm) {
  void *P = open_coll (_P, "w+");
  void *A = open_coll (_A, "r+");
  void *B = open_coll (_B, "r+");
  if (strstr(prm,"sparse")) sparse_product (P, A, B);  
  else                      dense_product (P, A, B);
  free_coll(P); free_coll(A); free_coll(B);
  return 0;
}
  
#define arg(i) ((i < argc) ? argv[i] : NULL)
#define a(i) ((i < argc) ? argv[i] : ((char*)""))

int main (int argc, char *argv[]) {
  // cumtx P = A x C [prm]
  if ((a(2)[0] == '=') && (a(4)[0] == 'x'))
    return gpu_product (arg(1), arg(3), arg(5), a(6));
  gpu_ram(0);
  return 0;
}

/*
#include <cuda_runtime.h>
#include "cublas_v2.h"
  cublasHandle_t H;
  cublasStatus_t ok, OK = CUBLAS_STATUS_SUCCESS;
  ok = cublasCreate(&H);
  if (ok != OK) { printf ("[%d] faild %s %d %d\n", ok, "cublasCreate", CUBLAS_STATUS_NOT_INITIALIZED, CUBLAS_STATUS_ALLOC_FAILED);  return 1; }
  ok = cublasGetVersion(H, &version);
  if (ok != OK) { printf ("[%d] faild %s\n", ok, "cublasVersion"); return 1; }
  ok = cublasDestroy(H);
  if (ok != OK) { printf ("[%d] faild %s\n", ok, "cublasDestroy"); return 1; }
*/
